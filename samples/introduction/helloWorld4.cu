#include "hip/hip_runtime.h"

#include <cstdio>
#include <iomanip>
#include <iostream>

// __global keyword indicates this methods works on device
__global__ void hello_world_cuda(){	
	printf("%6d%16d%16d%16d%16d%16d\n", blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(){
	// 2 block, 10 threads
	printf("**** Hello CUDA **** \n");
	printf("blockIdx.x\tblockIdx.y\tblockIdx.z\tthreadIdx.x\tthreadIdx.y\tthreadIdx.z\n");
	dim3 gridDim(2,2,2);
	dim3 blockDim(2,2,2);
	printf("blockIdx.x\tthreadIdx.x\tthreadIdx.y\tthreadIdx.z\n");
	hello_world_cuda <<< gridDim, blockDim >>>();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}