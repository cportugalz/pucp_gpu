#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipStream_t stream = NULL;

	const int m = 3;   /* 1 <= m <= 32 */
	const int n = 3;   /* 1 <= n <= 32 */
	const int lda = m; /* lda >= m */


	// const std::vector<hipDoubleComplex> A = {
	// 	make_hipDoubleComplex(4.024018e-12, 0),
	// 	make_hipDoubleComplex(1.139741e-12,-1.362755e-11),
	// 	make_hipDoubleComplex(-1.249698e-12,-1.221509e-11),
	// 	make_hipDoubleComplex(1.139741e-12, 1.362755e-11 ),
	// 	make_hipDoubleComplex(6.891594e-11, 2.524355e-29),
	// 	make_hipDoubleComplex(5.948327e-11, 2.556170e-13),
	// 	make_hipDoubleComplex(-1.249698e-12, 1.221509e-11),
	// 	make_hipDoubleComplex(5.948327e-11, -2.556170e-13),
	// 	make_hipDoubleComplex(5.587288e-11, 0.000000e+00)
	// };
	const std::vector<hipDoubleComplex> A = {
		make_hipDoubleComplex(2.068427e-12, 0.000000e+00),
		make_hipDoubleComplex(5.698705e-13, -6.813775e-12),
		make_hipDoubleComplex(-6.248490e-13, -6.107545e-12 ),
		make_hipDoubleComplex(5.698705e-13, 6.813775e-12),
		make_hipDoubleComplex(3.445797e-11, 1.262177e-29),
		make_hipDoubleComplex( 2.974163e-11,1.278085e-13),
		make_hipDoubleComplex(-6.248490e-13, 6.107545e-12),
		make_hipDoubleComplex(2.974163e-11, -1.278085e-13),
		make_hipDoubleComplex(2.793644e-11, 0.000000e+00)
	};
	std::vector<hipDoubleComplex> U(lda * m, {0,0});  /* m-by-m unitary matrix, left singular vectors  */
	std::vector<hipDoubleComplex> VT(lda * n, {0,0}); /* n-by-n unitary matrix, right singular vectors */
	std::vector<double> S(n, 0.0);        /* numerical singular value */
	std::vector<double> S_exact = {7.065283497082729,
								   1.040081297712078}; /* exact singular values */
	int info_gpu = 0;                                  /* host copy of error info */

	hipDoubleComplex *d_A = nullptr;
	double *d_S = nullptr;  /* singular values */
	hipDoubleComplex *d_U = nullptr;  /* left singular vectors */
	hipDoubleComplex *d_VT = nullptr; /* right singular vectors */
	hipDoubleComplex *d_W = nullptr;  /* W = S*VT */

	int *devInfo = nullptr;

	int lwork = 0; /* size of workspace */
	hipDoubleComplex *d_work = nullptr;
	double *d_rwork = nullptr;

	const double h_one = 1;
	const double h_minus_one = -1;

	std::printf("A = (matlab base-1)\n");
	print_matrix(m, n, A.data(), lda);
	std::printf("=====\n");

	/* step 1: create cusolver handle, bind a stream */
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
	CUBLAS_CHECK(hipblasCreate(&cublasH));

	CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

	/* step 2: copy A to device */
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * A.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(hipDoubleComplex) * U.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VT), sizeof(hipDoubleComplex) * VT.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(hipDoubleComplex) * lda * n));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&devInfo), sizeof(int)));

	CUDA_CHECK(
		hipMemcpyAsync(d_A, A.data(), sizeof(hipDoubleComplex) * A.size(), hipMemcpyHostToDevice, stream));

	/* step 3: query working space of SVD */
	CUSOLVER_CHECK(hipsolverDnZgesvd_bufferSize(cusolverH, m, n, &lwork));

	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(hipDoubleComplex) * lwork));

	/* step 4: compute SVD*/
	signed char jobu = 'N';  // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	CUSOLVER_CHECK(hipsolverDnZgesvd(cusolverH, jobu, jobvt, m, n, d_A, lda, d_S, d_U,
									lda, // ldu
									d_VT,
									lda, // ldvt,
									d_work, lwork, d_rwork, devInfo));

	CUDA_CHECK(
		hipMemcpyAsync(U.data(), d_U, sizeof(double) * U.size(), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(VT.data(), d_VT, sizeof(hipDoubleComplex) * VT.size(), hipMemcpyDeviceToHost,
							   stream));
	CUDA_CHECK(
		hipMemcpyAsync(S.data(), d_S, sizeof(double) * S.size(), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	std::printf("after gesvd: info_gpu = %d\n", info_gpu);
	if (0 == info_gpu) {
		std::printf("gesvd converges \n");
	} else if (0 > info_gpu) {
		std::printf("%d-th parameter is wrong \n", -info_gpu);
		exit(1);
	} else {
		std::printf("WARNING: info = %d : gesvd does not converge \n", info_gpu);
	}

	std::printf("S = singular values (matlab base-1)\n");
	print_matrix(n, 1, S.data(), n);
	std::printf("=====\n");

	std::printf("U = left singular vectors (matlab base-1)\n");
	print_matrix(m, m, U.data(), lda);
	std::printf("=====\n");

	std::printf("VT = right singular vectors (matlab base-1)\n");
	print_matrix(n, n, VT.data(), lda);
	std::printf("=====\n");

	// step 5: measure error of singular value
	// double ds_sup = 0;
	// for (int j = 0; j < n; j++) {
	// 	double err = fabs(S[j] - S_exact[j]);
	// 	ds_sup = (ds_sup > err) ? ds_sup : err;
	// }
	// std::printf("|S - S_exact| = %E \n", ds_sup);

	// CUBLAS_CHECK(hipblasZdgmm(cublasH, HIPBLAS_SIDE_LEFT, n, n, d_VT, lda, d_S, 1, d_W, lda));

	// CUDA_CHECK(
	// 	hipMemcpyAsync(d_A, A.data(), sizeof(double) * lda * n, hipMemcpyHostToDevice, stream));

	// CUBLAS_CHECK(hipblasDgemm(cublasH,
	// 						 HIPBLAS_OP_N,  // U
	// 						 HIPBLAS_OP_N,  // W
	// 						 m,            // number of rows of A
	// 						 n,            // number of columns of A
	// 						 n,            // number of columns of U
	// 						 &h_minus_one, /* host pointer */
	// 						 d_U,          // U
	// 						 lda,
	// 						 d_W,         // W
	// 						 lda, &h_one, /* hostpointer */
	// 						 d_A, lda));

	// double dR_fro = 0.0;
	// CUBLAS_CHECK(hipblasDnrm2(cublasH, lda * n, d_A, 1, &dR_fro));

	// std::printf("|A - U*S*VT| = %E \n", dR_fro);

	/* free resources */
	CUDA_CHECK(hipFree(d_A));
	CUDA_CHECK(hipFree(d_U));
	CUDA_CHECK(hipFree(d_VT));
	CUDA_CHECK(hipFree(d_S));
	CUDA_CHECK(hipFree(d_W));
	CUDA_CHECK(hipFree(devInfo));
	CUDA_CHECK(hipFree(d_work));
	CUDA_CHECK(hipFree(d_rwork));

	CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
	CUBLAS_CHECK(hipblasDestroy(cublasH));

	CUDA_CHECK(hipStreamDestroy(stream));

	CUDA_CHECK(hipDeviceReset());

	return EXIT_SUCCESS;
}
