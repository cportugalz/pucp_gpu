#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream = NULL;
	hipsolverGesvdjInfo_t gesvdj_params = NULL;

	const int m = 3;   /* 1 <= m <= 32 */
	const int n = 3;   /* 1 <= n <= 32 */
	const int lda = m; /* lda >= m */
	const int ldu = m; /* ldu >= m */
	const int ldv = n; /* ldv >= n */
	const int batchSize = 2;
	const int minmn = (m < n) ? m : n; /* min(m,n) */

	/*
	 *        |  1  -1  |
	 *   A0 = | -1   2  |
	 *        |  0   0  |
	 *
	 *   A0 = U0 * S0 * V0**T
	 *   S0 = diag(2.6180, 0.382)
	 *
	 *        |  3   4  |
	 *   A1 = |  4   7  |
	 *        |  0   0  |
	 *
	 *   A1 = U1 * S1 * V1**T
	 *   S1 = diag(9.4721, 0.5279)
	 */

	// std::vector<hipDoubleComplex> A(lda * n * batchSize, {0,0}); /* A = [A0 ; A1] */
	// std::vector<hipDoubleComplex> U(ldu * m * batchSize, {0,0}); /* U = [U0 ; U1] */
	// std::vector<hipDoubleComplex> V(ldv * n * batchSize, {0,0}); /* V = [V0 ; V1] */
	// std::vector<double> S(minmn * batchSize, 0);   /* S = [S0 ; S1] */
	// std::vector<int> info(batchSize, 0);             /* info = [info0 ; info1] */
	hipDoubleComplex* A = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex) * lda * n * batchSize); /* A = [A0 ; A1] */
	hipDoubleComplex* U = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex) * ldu * m * batchSize); /* U = [U0 ; U1] */
	hipDoubleComplex* V = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex) * ldv * n * batchSize); /* V = [V0 ; V1] */
	double* S =  (double*) malloc(sizeof(hipDoubleComplex) * minmn * batchSize);   /* S = [S0 ; S1] */
	int* info =  (int*) malloc(sizeof(int) *(batchSize) );             /* info = [info0 ; info1] */
	hipDoubleComplex *d_A = nullptr; /* lda-by-n-by-batchSize */
	hipDoubleComplex *d_U = nullptr; /* ldu-by-m-by-batchSize */
	hipDoubleComplex *d_V = nullptr; /* ldv-by-n-by-batchSize */
	double *d_S = nullptr; /* minmn-by-batchSize */
	int *d_info = nullptr; /* batchSize */

	int lwork = 0;            /* size of workspace */
	hipDoubleComplex *d_work = nullptr; /* device workspace for getrf */

	const double tol = 1.e-7;
	const int max_sweeps = 15;
	const int sort_svd = 0;                                  /* don't sort singular values */
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */

	hipDoubleComplex *A0 = A;
	hipDoubleComplex *A1 = A + lda * n; /* Aj is m-by-n */

	/*
	 *        |  1  -1  |
	 *   A0 = | -1   2  |
	 *        |  0   0  |
	 *   A0 is column-major
	 */
	A0[0 + 0 * lda] = make_hipDoubleComplex(4.024018e-12, 0);
	A0[1 + 0 * lda] = make_hipDoubleComplex(1.139741e-12,-1.362755e-11);
	A0[2 + 0 * lda] = make_hipDoubleComplex(-1.249698e-12,-1.221509e-11);
	A0[0 + 1 * lda] = make_hipDoubleComplex(1.139741e-12, 1.362755e-11 );
	A0[1 + 1 * lda] = make_hipDoubleComplex(6.891594e-11, 2.524355e-29);
	A0[2 + 1 * lda] = make_hipDoubleComplex(5.948327e-11, 2.556170e-13);
	A0[0 + 2 * lda] = make_hipDoubleComplex(-1.249698e-12, 1.221509e-11);
	A0[1 + 2 * lda] = make_hipDoubleComplex(5.948327e-11, -2.556170e-13);
	A0[2 + 2 * lda] = make_hipDoubleComplex(5.587288e-11, 0.000000e+00);

	/*
	 *        |  3   4  |
	 *   A1 = |  4   7  |
	 *        |  0   0  |
	 *   A1 is column-major
	 */
	A1[0 + 0 * lda] = make_hipDoubleComplex(2.068427e-12, 0.000000e+00);
	A1[1 + 0 * lda] = make_hipDoubleComplex(5.698705e-13, -6.813775e-12);
	A1[2 + 0 * lda] = make_hipDoubleComplex(-6.248490e-13, -6.107545e-12 );
	A1[0 + 1 * lda] = make_hipDoubleComplex(5.698705e-13, 6.813775e-12);
	A1[1 + 1 * lda] = make_hipDoubleComplex(3.445797e-11, 1.262177e-29);
	A1[2 + 1 * lda] = make_hipDoubleComplex( 2.974163e-11,1.278085e-13);
	A1[0 + 2 * lda] = make_hipDoubleComplex(-6.248490e-13, 6.107545e-12);
	A1[1 + 2 * lda] = make_hipDoubleComplex(2.974163e-11, -1.278085e-13);
	A1[2 + 2 * lda] = make_hipDoubleComplex(2.793644e-11, 0.000000e+00);

	std::printf("m = %d, n = %d \n", m, n);
	std::printf("tol = %E, default value is machine zero \n", tol);
	std::printf("max. sweeps = %d, default value is 100\n", max_sweeps);

	std::printf("A0 = (matlab base-1)\n");
	print_matrix(m, n, A, lda);
	std::printf("=====\n");

	std::printf("A1 = (matlab base-1)\n");
	print_matrix(m, n, A + lda * n, lda);
	std::printf("=====\n");

	/* step 1: create cusolver handle, bind a stream */
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

	CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

	/* step 2: configuration of gesvdj */
	CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

	/* default value of tolerance is machine zero */
	// CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));

	/* default value of max. sweeps is 100 */
	CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps));

	/* disable sorting */
	// CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd));

	/* step 3: copy A to device */
	// CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * A.size()));
	// CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(hipDoubleComplex) * U.size()));
	// CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(hipDoubleComplex) * V.size()));
	// CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
	// CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * lda * n * batchSize));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(hipDoubleComplex) * ldu * n * batchSize));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(hipDoubleComplex) * ldv * n * batchSize));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * minmn * batchSize));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * batchSize));
	CUDA_CHECK(
		hipMemcpyAsync(d_A, A, sizeof(hipDoubleComplex) * lda * n * batchSize, hipMemcpyHostToDevice, stream));

	/* step 4: query working space of gesvdjBatched */
	CUSOLVER_CHECK(hipsolverDnZgesvdjBatched_bufferSize(cusolverH, jobz, m, n, d_A, lda, d_S, d_U,
													   ldu, d_V, ldv, &lwork, gesvdj_params,
													   batchSize));

	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(hipDoubleComplex) * lwork));

	/* step 5: compute singular values of A0 and A1 */
	CUSOLVER_CHECK(hipsolverDnZgesvdjBatched(cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V,
											ldv, d_work, lwork, d_info, gesvdj_params, batchSize));

	CUDA_CHECK(
		hipMemcpyAsync(U, d_U, sizeof(hipDoubleComplex) * ldu * n * batchSize, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(
		hipMemcpyAsync(V, d_V, sizeof(hipDoubleComplex) * ldv * n * batchSize, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(
		hipMemcpyAsync(S, d_S, sizeof(double) *  minmn * batchSize, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(info, d_info, sizeof(int) * batchSize,hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	for (int i = 0; i < batchSize; i++) {
		if (0 == info[i]) {
			std::printf("matrix %d: gesvdj converges \n", i);
		} else if (0 > info[i]) {
			/* only info[0] shows if some input parameter is wrong.
			 * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
			 */
			std::printf("Error: %d-th parameter is wrong \n", -info[i]);
			exit(1);
		} else { /* info = m+1 */
				 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
			std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
		}
	}

	/* Step 6: show singular values and singular vectors */
	double *S0 = S;
	double *S1 = S + minmn;
	std::printf("==== \n");
	for (int i = 0; i < minmn; i++) {
		std::printf("S0(%d) = %20.16E\n", i + 1, S0[i]);
	}
	std::printf("==== \n");
	for (int i = 0; i < minmn; i++) {
		std::printf("S1(%d) = %20.16E\n", i + 1, S1[i]);
	}
	std::printf("==== \n");

	hipDoubleComplex *U0 = U;
	hipDoubleComplex *U1 = U + ldu * m; /* Uj is m-by-m */
	std::printf("U0 = (matlab base-1)\n");
	print_matrix(m, m, U0, ldu);
	std::printf("U1 = (matlab base-1)\n");
	print_matrix(m, m, U1, ldu);

	hipDoubleComplex *V0 = V;
	hipDoubleComplex *V1 = V + ldv * n; /* Vj is n-by-n */
	std::printf("V0 = (matlab base-1)\n");
	print_matrix(n, n, V0, ldv);
	std::printf("V1 = (matlab base-1)\n");
	print_matrix(n, n, V1, ldv);

	/* free resources */
	CUDA_CHECK(hipFree(d_A));
	CUDA_CHECK(hipFree(d_U));
	CUDA_CHECK(hipFree(d_V));
	CUDA_CHECK(hipFree(d_S));
	CUDA_CHECK(hipFree(d_info));
	CUDA_CHECK(hipFree(d_work));

	CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));

	CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

	CUDA_CHECK(hipStreamDestroy(stream));

	CUDA_CHECK(hipDeviceReset());

	return EXIT_SUCCESS;
}
