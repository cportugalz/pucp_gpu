#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream = NULL;
	hipsolverSyevjInfo_t syevj_params = NULL;

	const int m = 3;
	const int lda = m;
	const int batchSize = 2;
	/*
	 *        |  1  -1   0 |
	 *   A0 = | -1   2   0 |
	 *        |  0   0   0 |
	 *
	 *   A0 = V0 * W0 * V0**T
	 *
	 *   W0 = diag(0, 0.3820, 2.6180)
	 *
	 *        |  3   4  0 |
	 *   A1 = |  4   7  0 |
	 *        |  0   0  0 |
	 *
	 *   A1 = V1 * W1 * V1**T
	 *
	 *   W1 = diag(0, 0.5279, 9.4721)
	 *
	 */

	std::vector<hipDoubleComplex> A(lda * m * batchSize, {0,0}); /* V = [A0 ; A1] */
	std::vector<hipDoubleComplex> V(lda * m * batchSize, {0,0}); /* V = [V0 ; V1] */
	std::vector<double> W(m * batchSize, 0);       /* W = [W0 ; W1] */
	std::vector<int> info(batchSize, 0);           /* info = [info0 ; info1] */

	hipDoubleComplex *d_A = nullptr;    /* lda-by-m-by-batchSize */
	double *d_W = nullptr;    /* m-by-batchSize */
	int *d_info = nullptr;    /* batchSize */
	hipDoubleComplex *d_work = nullptr; /* device workspace for syevjBatched */
	int lwork = 0;            /* size of workspace */

	/* configuration of syevj  */
	const double tol = 1.e-9;
	const int max_sweeps = 100;
	const int sort_eig = 1;                                  /* don't sort eigenvalues */
	const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
	const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

	hipDoubleComplex *A0 = A.data();
	hipDoubleComplex *A1 = A.data() + lda * m;
	/*
	 *        |  1  -1   0 |
	 *   A0 = | -1   2   0 |
	 *        |  0   0   0 |
	 *   A0 is column-major
	 */
			 
		 
	   
	A0[0 + 0 * lda] = make_hipDoubleComplex(4.024018e-12, 0);
	A0[1 + 0 * lda] = make_hipDoubleComplex(1.139741e-12,-1.362755e-11);
	A0[2 + 0 * lda] = make_hipDoubleComplex(-1.249698e-12,-1.221509e-11);
	A0[0 + 1 * lda] = make_hipDoubleComplex(1.139741e-12, 1.362755e-11 );
	A0[1 + 1 * lda] = make_hipDoubleComplex(6.891594e-11, 2.524355e-29);
	A0[2 + 1 * lda] = make_hipDoubleComplex(5.948327e-11, 2.556170e-13);
	A0[0 + 2 * lda] = make_hipDoubleComplex(-1.249698e-12, 1.221509e-11);
	A0[1 + 2 * lda] = make_hipDoubleComplex(5.948327e-11, -2.556170e-13);
	A0[2 + 2 * lda] = make_hipDoubleComplex(5.587288e-11, 0.000000e+00);
	// A0[0 + 0 * lda] = make_hipDoubleComplex(2,0);
	// A0[1 + 0 * lda] = make_hipDoubleComplex(0,1);
	// A0[2 + 0 * lda] = make_hipDoubleComplex(0,0);
	// A0[0 + 1 * lda] = make_hipDoubleComplex(0,1);
	// A0[1 + 1 * lda] = make_hipDoubleComplex(1,0);
	// A0[2 + 1 * lda] = make_hipDoubleComplex(0,5);
	// A0[0 + 2 * lda] = make_hipDoubleComplex(0,0);
	// A0[1 + 2 * lda] = make_hipDoubleComplex(0,5);
	// A0[2 + 2 * lda] = make_hipDoubleComplex(4,0);
	/*
	 *        |  3   4  0 |
	 *   A1 = |  4   7  0 |
	 *        |  0   0  0 |
	 *   A1 is column-major
	 */
			   
	A1[0 + 0 * lda] = make_hipDoubleComplex(2.068427e-12, 0.000000e+00);
	A1[1 + 0 * lda] = make_hipDoubleComplex(5.698705e-13, -6.813775e-12);
	A1[2 + 0 * lda] = make_hipDoubleComplex(-6.248490e-13, -6.107545e-12 );
	A1[0 + 1 * lda] = make_hipDoubleComplex(5.698705e-13, 6.813775e-12);
	A1[1 + 1 * lda] = make_hipDoubleComplex(3.445797e-11, 1.262177e-29);
	A1[2 + 1 * lda] = make_hipDoubleComplex( 2.974163e-11,1.278085e-13);
	A1[0 + 2 * lda] = make_hipDoubleComplex(-6.248490e-13, 6.107545e-12);
	A1[1 + 2 * lda] = make_hipDoubleComplex(2.974163e-11, -1.278085e-13);
	A1[2 + 2 * lda] = make_hipDoubleComplex(2.793644e-11, 0.000000e+00);
	// A1[0 + 0 * lda] = make_hipDoubleComplex(0,2);
	// A1[1 + 0 * lda] = make_hipDoubleComplex(0,-1);
	// A1[2 + 0 * lda] = make_hipDoubleComplex(0,0);
	// A1[0 + 1 * lda] = make_hipDoubleComplex(0,1);
	// A1[1 + 1 * lda] = make_hipDoubleComplex(1,0);
	// A1[2 + 1 * lda] = make_hipDoubleComplex(0,-5);
	// A1[0 + 2 * lda] = make_hipDoubleComplex(0,0);
	// A1[1 + 2 * lda] = make_hipDoubleComplex(0,5);
	// A1[2 + 2 * lda] = make_hipDoubleComplex(0,4);

	std::printf("A0 = (matlab base-1)\n");
	print_matrix(m, m, A0, lda);
	std::printf("=====\n");

	std::printf("A1 = (matlab base-1)\n");
	print_matrix(m, m, A1, lda);
	std::printf("=====\n");

	/* step 1: create cusolver handle, bind a stream */
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

	CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

	/* step 2: configuration of syevj */
	CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

	/* default value of tolerance is machine zero */
	CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

	/* default value of max. sweeps is 100 */
	CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

	/* disable sorting */
	CUSOLVER_CHECK(hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig));

	/* step 3: copy A to device */
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * W.size()));
	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));

	CUDA_CHECK(
		hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
	/* step 4: query working space of syevj */
	CUSOLVER_CHECK(hipsolverDnZheevjBatched_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W,
													  &lwork, syevj_params, batchSize));

	CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(hipDoubleComplex) * lwork));

	/* step 5: compute eigen-pair   */
	CUSOLVER_CHECK(hipsolverDnZheevjBatched(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork,
										   d_info, syevj_params, batchSize));

	CUDA_CHECK(
		hipMemcpyAsync(V.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(
		hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(info.data(), d_info, sizeof(int) * info.size(),
							   hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	for (int i = 0; i < batchSize; i++) {
		if (0 == info[i]) {
			std::printf("matrix %d: syevj converges \n", i);
		} else if (0 > info[i]) {
			/* only info[0] shows if some input parameter is wrong.
			 * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
			 */
			std::printf("Error: %d-th parameter is wrong \n", -info[i]);
			exit(1);
		} else { /* info = m+1 */
				 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
			std::printf("WARNING: matrix %d, info = %d : sygvj does not converge \n", i, info[i]);
		}
	}

	/* Step 6: show eigenvalues and eigenvectors */
	double *W0 = W.data();
	double *W1 = W.data() + m;

	std::printf("==== \n");
	for (int i = 0; i < m; i++) {
		std::printf("W0[%d] = %e \n", i, W0[i]);
	}
	std::printf("==== \n");
	for (int i = 0; i < m; i++) {
		std::printf("W1[%d] = %e \n", i, W1[i]);
	}
	std::printf("==== \n");

	hipDoubleComplex *V0 = V.data();
	hipDoubleComplex *V1 = V.data() + lda * m;

	std::printf("V0 = (matlab base-1)\n");
	print_matrix(m, m, V0, lda);
	std::printf("V1 = (matlab base-1)\n");
	print_matrix(m, m, V1, lda);

	/* free resources */
	CUDA_CHECK(hipFree(d_A));
	CUDA_CHECK(hipFree(d_W));
	CUDA_CHECK(hipFree(d_info));
	CUDA_CHECK(hipFree(d_work));

	CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));

	CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

	CUDA_CHECK(hipStreamDestroy(stream));

	CUDA_CHECK(hipDeviceReset());

	return EXIT_SUCCESS;
}
