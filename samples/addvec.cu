#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__
void vecAddKernel(float* A,float* B, float* C,int n){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<n) C[i] = A[i] + B[i];
}

int main(){
	int device;
	hipGetDevice(&device);

	struct hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device);

	cout << device;

	

	int n = 100;
	float *A,*B,*C;
	A = new float[n];
	B = new float[n];
	C = new float[n];
	for(int i = 0; i < n; ++i)
		A[i] = B[i] = i;
	float *d_A,*d_B,*d_C;
	int size = n * sizeof(float);
	hipMalloc((void**)&d_A,size);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	vecAddKernel<<<std::ceil(n/20),20>>>(d_A,d_B,d_C,n);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	for(int i = 0; i < n; ++i)
		std::cout << C[i] << " ";
	std::cout << std::endl;
	delete A,B,C;
	return 0;
}
