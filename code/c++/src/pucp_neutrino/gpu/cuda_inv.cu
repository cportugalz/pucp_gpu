#include "hip/hip_runtime.h"
#include "probabilities.h"
#include "hutils.h"
#include "dutils.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <time.h>
using namespace std;

// GPU Kernel for visible decay
__global__ void gpu_invisible_decay(
	hipDoubleComplex* _U, int _size_data, int _sigN, double _L, double _rho,
	double* _dm, double* _alpha, hipDoubleComplex** _batchedDM,
	hipDoubleComplex** _batchedPot, hipDoubleComplex** _batchedU){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < _size_data){
		//Copying _size_data of U to batchedU
		for (int i=0; i<9; i++) {
			_batchedU[tid][i] = _U[i];
		}
		double energy = (tid+1)/100.0 * 1e9;
		// printf("Calling threadIdx: %d  for energy %e\n", tid, energy);
		double rho = _sigN * _rho;
		// Matriz de masas y Decay
		_batchedDM[tid][0] = {0, -0.5 * _alpha[0] / energy};
		_batchedDM[tid][4] = {0.5 * _dm[0] / energy, -0.5 * _alpha[1] / energy};
		_batchedDM[tid][8] = {0.5 * _dm[1] / energy, -0.5 * _alpha[2] / energy};

		_batchedPot[tid][0] = {rho * 7.63247 * 0.5 * 1.e-14, 0}; _batchedPot[tid][3] = _batchedDM[tid][3];
		_batchedPot[tid][6]= _batchedDM[tid][6]; _batchedPot[tid][1] = _batchedDM[tid][1]; _batchedPot[tid][4] = _batchedDM[tid][0];
		_batchedPot[tid][7] = _batchedDM[tid][7]; _batchedPot[tid][2] = _batchedDM[tid][2]; _batchedPot[tid][5] = _batchedDM[tid][5];
		_batchedPot[tid][8] = _batchedDM[tid][0];
	}
}


__global__ void sum_batched(
	hipDoubleComplex** Pot, hipDoubleComplex** Hff2, hipDoubleComplex** Hff3, hipDoubleComplex* d_A, int _batch_count) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	//   0		1
	// 1 4 7	4 7 10
	// 2 5 8    5 8 11
	// 3 6 9    6 9 12
	// Pot =  { { 1,2,3,4,5,6,7,8,9}, {4,5,6,7,8,9,10,11,12}, {}, {} }
	// Hff2 = { { 1,2,3,4,5,6,7,8,9}, {4,5,6,7,8,9,10,11,12}, {}, {} }
	// Hff3 = { { 2,4,6,8, 10 ...  }, {...}, { ...}, { ... } }
	if (tid < _batch_count) {
		for (int i=0; i<9; i++) {
				Hff3[tid][i].x = Hff2[tid][i].x + Pot[tid][i].x;
				Hff3[tid][i].y = Hff2[tid][i].y + Pot[tid][i].y;
				d_A[tid * 9 + i].x =  Hff3[tid][i].x;
				d_A[tid * 9 + i].y =  Hff3[tid][i].y;
				// (d_A + tid * 9 + i)->x =  12;
				// (d_A + tid * 9 + i)->y =  10;
				// printf("[%d ]%e %ej \n", tid * 9 + i, d_A[tid * 9 + i].x, d_A[tid * 9 + i].y);
		}
	}
}


__global__ void building_SandV(
	hipDoubleComplex ** _batchedS, hipDoubleComplex** _batchedDM, double* _d_S,
	hipDoubleComplex* _d_V,  hipDoubleComplex** _batchedV, int _batch_count,
	hipDoubleComplex _I, double _GevkmToevsq, double _L){
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < _batch_count) {
			hipDoubleComplex _minusI = make_hipDoubleComplex(-1*_I.x, -1*_I.y);
			double L = _L * 1e9/_GevkmToevsq;
			hipDoubleComplex eigv0 = make_hipDoubleComplex(_d_S[tid*3]*_minusI.x, _d_S[tid*3]*_minusI.y);
			hipDoubleComplex eigv0xL = make_hipDoubleComplex(L*eigv0.x, L*eigv0.y);
			hipDoubleComplex eigv1 = make_hipDoubleComplex(_d_S[tid*3+1]*_minusI.x, _d_S[tid*3+1]*_minusI.y);
			hipDoubleComplex eigv1xL = make_hipDoubleComplex(L*eigv1.x, L * eigv1.y);
			hipDoubleComplex eigv2 = make_hipDoubleComplex(_d_S[tid*3+2]*_minusI.x, _d_S[tid*3+2]*_minusI.y);
			hipDoubleComplex eigv2xL = make_hipDoubleComplex(L*eigv2.x, L*eigv2.y);
			_batchedS[tid][0] = exp(eigv0xL);
			_batchedS[tid][4] = exp(eigv1xL);
			_batchedS[tid][8] = exp(eigv2xL);
			_batchedS[tid][1] = _batchedDM[tid][0];
			_batchedS[tid][2] = _batchedDM[tid][0];
			_batchedS[tid][3] = _batchedDM[tid][0];
			_batchedS[tid][5] = _batchedDM[tid][0];
			_batchedS[tid][6] = _batchedDM[tid][0];
			_batchedS[tid][7] = _batchedDM[tid][0];
			for (int iter=0; iter<9; iter++) {
				_batchedV[tid][iter].x = _d_V[iter + tid*9].x;
				_batchedV[tid][iter].y = _d_V[iter + tid*9].y;
			}
		}
	}


__global__ void buildP(double** _batchedP, hipDoubleComplex** _batchedS, int _batch_count){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < _batch_count){
		for (int i=0; i<3; i++){
			for (int j=0; j<3; j++){
				hipDoubleComplex N = hipCmul(_batchedS[tid][i*3+j], _batchedS[tid][i*3+j]);
				_batchedP[tid][i+j*3] = hipCabs(N);
			}
		}
	}
}


// GPU Invisible Decay
void cuda_InvisibleDecay(
	hipDoubleComplex* _U, int _batch_count, int _sigN, double _L, double _rho,
	double* _dm, double* _alpha) {
		float threads = 1024;
		using data_type =  hipDoubleComplex;
		const int m = 3;
		const int n = 3;
		const int k = 3;
		const int lda = 3;
		const int ldb = 3;
		const int ldc = 3;
		const int ldu = m; /* ldu >= m */
		const int ldv = n; /* ldv >= n */
		const int minmn = (m < n) ? m : n; /* min(m,n) */

		int blocks = ceil(_batch_count/threads);
		printf("Assigning host memory for Invisible decay.\n");

		// Assigning memory to batched matrices of mass and decay
		data_type** batchedU = nullptr;
		data_type* host_batchedU[_batch_count];
		data_type* device_batchedU[_batch_count];

		data_type** batchedDM = nullptr;
		data_type* host_batchedDM[_batch_count];
		data_type* device_batchedDM[_batch_count];

		data_type** batchedPot = nullptr;
		data_type* host_batchedPot[_batch_count];
		data_type* device_batchedPot[_batch_count];

		data_type** batchedHff = nullptr;
		data_type* host_batchedHff[_batch_count];
		data_type* device_batchedHff[_batch_count];

		data_type** batchedHff2 = nullptr;
		data_type* host_batchedHff2[_batch_count];
		data_type* device_batchedHff2[_batch_count];

		data_type** batchedHff3 = nullptr;
		data_type* host_batchedHff3[_batch_count];
		data_type* device_batchedHff3[_batch_count];

		data_type** batchedS = nullptr;
		data_type* host_batchedS[_batch_count];
		data_type* device_batchedS[_batch_count];

		data_type** batchedS1= nullptr;
		data_type* host_batchedS1[_batch_count];
		data_type* device_batchedS1[_batch_count];

		data_type** batchedS2 = nullptr;
		data_type* host_batchedS2[_batch_count];
		data_type* device_batchedS2[_batch_count];

		data_type** batchedV = nullptr;
		data_type* host_batchedV[_batch_count];
		data_type* device_batchedV[_batch_count];

		data_type** batchedInvV = nullptr;
		data_type* host_batchedInvV[_batch_count];
		data_type* device_batchedInvV[_batch_count];

		double** batchedP = nullptr;
		double* host_batchedP[_batch_count];
		double* device_batchedP[_batch_count];

		for (int i = 0; i < _batch_count; i++) {
			hipHostMalloc(&host_batchedU[i],   sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedDM[i],  sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedPot[i], sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedHff[i], sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedHff2[i],sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedHff3[i],sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedS[i] ,  sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedS1[i],  sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedS2[i],  sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedV[i],   sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedInvV[i],sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedP[i],   sizeof(double)*m*n);

			for (int j = 0; j < m*n; j++){
				host_batchedU[i][j]  = make_hipDoubleComplex(0.0,0.0);
				host_batchedDM[i][j]  = make_hipDoubleComplex(0.0,0.0);
				host_batchedPot[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedHff[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedHff2[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedHff3[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedS[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedS1[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedS2[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedV[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedInvV[i][j] = make_hipDoubleComplex(0.0,0.0);
				host_batchedP[i][j] = 0;
			}
		}
		printf("Assigning device memory for Invisible Decay.\n");
		for (int i = 0; i < _batch_count; i++) {
			hipMalloc(reinterpret_cast<void **>(&device_batchedU[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedDM[i]), 	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedPot[i]),	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedHff[i]), 	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedHff2[i]),	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedHff3[i]),	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedS[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedS1[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedS2[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedV[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedInvV[i]),	sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedP[i]),		sizeof(double) * m*n);
		}

		hipMalloc(reinterpret_cast<void **>(&batchedU),  	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedDM),  	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedPot), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedHff), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedHff2), _batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedHff3), _batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedS), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedS1), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedS2), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedV), 	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedInvV), _batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedP), 	_batch_count*sizeof(double *));
		printf("Copying host memory to device memory.\n");
		for (int i = 0; i < _batch_count; i++) {
			hipMemcpy(device_batchedU[i], host_batchedU[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedDM[i], host_batchedDM[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedPot[i], host_batchedPot[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedHff[i], host_batchedHff[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedHff2[i], host_batchedHff2[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedHff3[i], host_batchedHff3[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedS[i], host_batchedS[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedS1[i], host_batchedS1[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedS2[i], host_batchedS2[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedV[i], host_batchedV[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedInvV[i], host_batchedInvV[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedP[i], host_batchedP[i], sizeof(double) * m*n, hipMemcpyHostToDevice);
		}
		hipMemcpy(batchedU, device_batchedU, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedDM, device_batchedDM, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedPot, device_batchedPot, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedHff, device_batchedHff, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedHff2, device_batchedHff2, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedHff3, device_batchedHff3, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedS, device_batchedS, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedS1, device_batchedS1, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedS2, device_batchedS2, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedV, device_batchedV, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedInvV, device_batchedInvV, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedP, device_batchedP, sizeof(double*)*_batch_count, hipMemcpyHostToDevice);
		clock_t start_time = clock();
		printf("Calling Invisible Decay Kernel with %d blocks and %.0f threads per block\n", blocks, threads);
		gpu_invisible_decay<<<blocks, threads>>>(
			_U, _batch_count, _sigN, _L, _rho, _dm, _alpha,
			batchedDM, batchedPot, batchedU);
		hipDeviceSynchronize();
		// CuBlas Operations
		hipblasHandle_t cublasH = NULL;
		// hipStream_t stream = NULL;

		const data_type alpha = {1.0, 0.0};
		const data_type beta = {1.0, 0.0};

		hipblasCreate(&cublasH);
		// CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
		// hipblasSetStream(cublasH, stream));
		hipblasZgemmBatched(
			cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedU, lda,
			(const data_type*  const*) batchedDM, ldb, &beta, batchedHff, ldc, _batch_count);
		// CUDA_CHECK(hipStreamSynchronize(stream));
		hipblasZgemmBatched(
			cublasH, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, &alpha, (const data_type * const *)batchedHff, lda,
			(const data_type*  const*) batchedU, ldb, &beta, batchedHff2, ldc, _batch_count);
		// CUDA_CHECK(hipStreamSynchronize(stream));
		// hipblasDestroy(cublasH));

		// Calculating eigen value with cuSolver
		hipsolverHandle_t cusolverH = NULL;
		hipsolverGesvdjInfo_t gesvdj_params = NULL;
		data_type *d_A = nullptr;    /* lda-by-m-by-batchSize */
		data_type *h_A = (data_type*) malloc(sizeof(data_type) * _batch_count * m * n);    /* lda-by-m-by-batchSize */
		data_type *d_U = nullptr;    /* lda-by-m-by-batchSize */
		data_type *h_U = (data_type*) malloc( sizeof(data_type) * ldu * m * _batch_count);    /* lda-by-m-by-_batch_count */
		data_type *d_V = nullptr;    /* lda-by-m-by-_batch_count */
		data_type *h_V = (data_type*) malloc( sizeof(data_type) * ldv * n * _batch_count);    /* lda-by-m-by-_batch_count */
		double* S = (double*) malloc(sizeof(hipDoubleComplex) * minmn * _batch_count);
		double *d_S = nullptr; /* minmn-by-batchSize */
		int* info = (int*) malloc(sizeof(int) * _batch_count);
		int *d_info = nullptr; /* batchSize */


		int lwork = 0;            /* size of workspace */
		data_type *d_work = nullptr; /* device workspace for getrf */

		// const double tol = 1.e-7;
		// const int max_sweeps = 50;
		const int sort_svd = 0;                                  /* don't sort singular values */
		const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */
		hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * _batch_count * m * n);
		sum_batched<<<blocks, threads>>> (batchedPot, batchedHff2, batchedHff3, d_A, _batch_count);
		hipDeviceSynchronize();
		// Hff[0] = U[0] * DM[0] * UC[0] + Pot[0]
		// Hff[1] = U[1] * DM[1] * UC[1] + Pot[1]
		// Hff[N-1] = U[N-1] * DM[N-1] * UC[N-1] + Pot[N-1]
		/* step 1: create cusolver handle, bind a stream */
		hipsolverDnCreate(&cusolverH);
		// hipsolverSetStream(cusolverH, stream);

		/* step 2: configuration of syevj */
		hipsolverDnCreateGesvdjInfo(&gesvdj_params);

		/* default value of tolerance is machine zero */
		// hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol);

		/* default value of max. sweeps is 100 */
		// hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);


		/* disable sorting */
		hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd);
		hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(data_type) * ldu * m * _batch_count);
		hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(data_type) * ldv * n * _batch_count);
		hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * minmn * _batch_count);
		hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * _batch_count);

		hipsolverDnZgesvdjBatched_bufferSize(cusolverH, jobz, m, n, d_A, lda, d_S, d_U,
			ldu, d_V, ldv, &lwork, gesvdj_params, _batch_count);
		hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * lwork);

		hipsolverDnZgesvdjBatched(cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V,
			ldv, d_work, lwork, d_info, gesvdj_params, _batch_count);

		// building S from eigenvalues and ordering V to array of vectors
		building_SandV<<<blocks, threads>>> (
			batchedS, batchedDM, d_S, d_V, batchedV, _batch_count,
			make_hipDoubleComplex(ProbConst::I.real(), ProbConst::I.imag()),
			ProbConst::GevkmToevsq, _L);
		// building the inverse of V from batchedV
		
		cublasZmatinvBatched(
				cublasH, n, (const hipDoubleComplex * const *) batchedV, lda, batchedInvV, lda,
				d_info, _batch_count);
		// S = S*V * Vinv
		hipblasZgemmBatched(
			cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedV, lda,
			(const data_type*  const*) batchedS, ldb, &beta, batchedS1, ldc, _batch_count);
		hipblasZgemmBatched(
			cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedS1, lda,
			(const data_type*  const*) batchedInvV, ldb, &beta, batchedS2, ldc, _batch_count);
		buildP<<<blocks, threads>>>(batchedP, batchedS2, _batch_count);
		clock_t stop_time = clock();
		printf("Computation time: %.7fs\n", (double)(stop_time - start_time)/CLOCKS_PER_SEC);
		for (int i = 0; i < _batch_count; i++) {
			hipMemcpy(host_batchedU[i], device_batchedU[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedDM[i], device_batchedDM[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedPot[i], device_batchedPot[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedHff[i], device_batchedHff[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedHff2[i], device_batchedHff2[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedHff3[i], device_batchedHff3[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedS[i], device_batchedS[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedS2[i], device_batchedS2[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedV[i], device_batchedV[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedP[i], device_batchedP[i], sizeof(double)* m*n, hipMemcpyDeviceToHost );
		}
		// CUDA_CHECK(hipMemcpy(h_A, d_A, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost));
		hipMemcpy(h_U, d_U, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost);
		// hipMemcpy(h_V, d_V, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost));
		hipMemcpy(S, d_S, sizeof(double) * minmn * _batch_count, hipMemcpyDeviceToHost);
		hipMemcpy(info, d_info, sizeof(int) * _batch_count, hipMemcpyDeviceToHost);

		// hipStreamSynchronize(stream));
		hipDeviceSynchronize();

		for (int i=0; i < _batch_count; i++) {
			printf("U[%d]:\n",i);
			print_matrix(m, n, host_batchedU[i], lda);
			printf("DM[%d]:\n",i);
			print_matrix(m, n, host_batchedDM[i], lda);
			printf("Pot[%d]:\n",i);
			print_matrix(m, n, host_batchedPot[i], lda);
			printf("Hff[%d]:\n",i);
			print_matrix(m, n, host_batchedHff[i], lda);
			printf("Hff2[%d]:\n",i);
			print_matrix(m, n, host_batchedHff2[i], lda);
			printf("Hff3[%d]:\n",i);
			print_matrix(m, n, host_batchedHff3[i], lda);
			// printf("H_A[%d]:\n",i);
			// print_matrix(m, n, h_A +  m * lda * i , 3);
			std::printf("Eigen Values: \n");
			for (int v = 0; v < minmn; v++) {
				std::printf("S0(%d) = %e\n", v + 1, S[i * m + v]);
			}
			printf("Eigen Vectors:\n");
			print_matrix(m, m, h_V + i * m * lda, ldv);
			printf("S[%d]:\n", i);
			print_matrix(m, m, host_batchedS[i], lda);
			// printf("V[%d]:\n", i);
			// print_matrix(m, m, host_batchedV[i], lda);
			printf("S2[%d]:\n", i);
			print_matrix(m, m, host_batchedS2[i], lda);
			printf("P[%d]:\n", i);
			print_matrix(m, m, host_batchedP[i], lda);
		}
		for (int i = 0; i < _batch_count; i++) {
			hipHostFree(host_batchedU[i]);
			hipHostFree(host_batchedDM[i]);
			hipHostFree(host_batchedPot[i]);
			hipHostFree(host_batchedHff[i]);
			hipHostFree(host_batchedHff2[i]);
			hipHostFree(host_batchedHff3[i]);
			hipHostFree(host_batchedS[i]);
			hipHostFree(host_batchedS1[i]);
			hipHostFree(host_batchedS2[i]);
			hipHostFree(host_batchedV[i]);
			hipHostFree(host_batchedInvV[i]);
			hipHostFree(host_batchedP[i]);
			hipFree(device_batchedU[i]); 
			hipFree(device_batchedDM[i]);
			hipFree(device_batchedPot[i]);
			hipFree(device_batchedHff[i]);
			hipFree(device_batchedHff2[i]);
			hipFree(device_batchedHff3[i]);
			hipFree(device_batchedS[i]); 
			hipFree(device_batchedS1[i]);
			hipFree(device_batchedS2[i]);
			hipFree(device_batchedV[i]);
			hipFree(device_batchedInvV[i]);
			hipFree(device_batchedP[i]);
		}
		hipFree(batchedU);
		hipFree(batchedDM);
		hipFree(batchedPot);
		hipFree(batchedHff);
		hipFree(batchedHff2);
		hipFree(batchedHff3);
		hipFree(batchedS);
		hipFree(batchedS1);
		hipFree(batchedS2);
		hipFree(batchedV);
		hipFree(batchedInvV);
		hipFree(batchedP);

		
		hipFree(d_A);
		hipFree(d_U);
		hipFree(d_V);
		hipFree(d_S);
		hipFree(d_info);
		hipFree(d_work);

		hipsolverDnDestroyGesvdjInfo(gesvdj_params);
		hipsolverDnDestroy(cusolverH);
		// CUDA_CHECK(hipStreamDestroy(stream));
		hipDeviceReset();
}

