#include "hip/hip_runtime.h"
#include "cuda_probabilities.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "utils.h"
#include <hipsolver.h>
#include <time.h>
using namespace std;


// GPU Kernel for visible decay
__global__ void gpu_nsi(
	hipDoubleComplex* _U, int _size_data, int _sigN, double _L, double _rho,
	double* _dm, double* _alpha, double* _events, hipDoubleComplex** _batchedDM,
	hipDoubleComplex** _batchedPot, hipDoubleComplex** _batchedU){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < _size_data){
		//Copying _size_data of U to batchedU
		for (int i=0; i<9; i++) {
			_batchedU[tid][i] = _U[i];
		}
		double energy = (tid+1)/100.0 * 1e9;
		// printf("Calling threadIdx: %d  for energy %e\n", tid, energy);
		double rho = _sigN * _rho;
		// Matriz de masas y Decay
		_batchedDM[tid][4] = {0.5 * _dm[0] / energy, 0};
		_batchedDM[tid][8] = {0.5 * _dm[1] / energy, 0};

		_batchedPot[tid][0] = {rho * 7.63247 * 0.5 * 1.e-14, 0}; _batchedPot[tid][3] = _batchedDM[tid][3];
		_batchedPot[tid][6]= _batchedDM[tid][6]; _batchedPot[tid][1] = _batchedDM[tid][1]; _batchedPot[tid][4] = _batchedDM[tid][0];
		_batchedPot[tid][7] = _batchedDM[tid][7]; _batchedPot[tid][2] = _batchedDM[tid][2]; _batchedPot[tid][5] = _batchedDM[tid][5];
		_batchedPot[tid][8] = _batchedDM[tid][0];
	}
}


// GPU  Non Standard Interaction
void cuda_NonStandardInteraction(
	hipDoubleComplex* _U, int _batch_count, int _sigN, double _L, double _rho,
	double* _dm, double* _alpha, double* _events) {
		float threads = 1024;
		using data_type =  hipDoubleComplex;
		const int m = 3;
		const int n = 3;
		const int k = 3;
		const int lda = 3;
		const int ldb = 3;
		const int ldc = 3;
		const int ldu = m; /* ldu >= m */
		const int ldv = n; /* ldv >= n */
		const int minmn = (m < n) ? m : n; /* min(m,n) */

		int blocks = ceil(_batch_count/threads);
		printf("Assigning host memory for Non Standard Interaction.\n");

		// Assigning memory to batched matrices of mass and decay
		data_type** batchedU = nullptr;
		// data_type* host_batchedU[_batch_count];
		data_type* device_batchedU[_batch_count];

		data_type** batchedDM = nullptr;
		data_type* host_batchedDM[_batch_count];
		data_type* device_batchedDM[_batch_count];

		data_type** batchedPot = nullptr;
		data_type* host_batchedPot[_batch_count];
		data_type* device_batchedPot[_batch_count];

		// data_type** batchedHff = nullptr;
		// data_type* host_batchedHff[_batch_count];
		// data_type* device_batchedHff[_batch_count];

		// data_type** batchedHff2 = nullptr;
		// data_type* host_batchedHff2[_batch_count];
		// data_type* device_batchedHff2[_batch_count];

		// data_type** batchedHff3 = nullptr;
		// data_type* host_batchedHff3[_batch_count];
		// data_type* device_batchedHff3[_batch_count];

		// data_type** batchedS = nullptr;
		// data_type* host_batchedS[_batch_count];
		// data_type* device_batchedS[_batch_count];

		// data_type** batchedS1= nullptr;
		// data_type* host_batchedS1[_batch_count];
		// data_type* device_batchedS1[_batch_count];

		// data_type** batchedS2 = nullptr;
		// data_type* host_batchedS2[_batch_count];
		// data_type* device_batchedS2[_batch_count];

		// data_type** batchedV = nullptr;
		// data_type* host_batchedV[_batch_count];
		// data_type* device_batchedV[_batch_count];

		// data_type** batchedInvV = nullptr;
		// data_type* host_batchedInvV[_batch_count];
		// data_type* device_batchedInvV[_batch_count];

		// double** batchedP = nullptr;
		// double* host_batchedP[_batch_count];
		// double* device_batchedP[_batch_count];

		for (int i = 0; i < _batch_count; i++) {
			// hipHostMalloc(&host_batchedU[i],   sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedDM[i],  sizeof(data_type)*m*n);
			hipHostMalloc(&host_batchedPot[i], sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedHff[i], sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedHff2[i],sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedHff3[i],sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedS[i] ,  sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedS1[i],  sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedS2[i],  sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedV[i],   sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedInvV[i],sizeof(data_type)*m*n);
			// hipHostMalloc(&host_batchedP[i],   sizeof(double)*m*n);

			for (int j = 0; j < m*n; j++){
				// host_batchedU[i][j]  = make_hipDoubleComplex(0.0,0.0);
				host_batchedDM[i][j]  = make_hipDoubleComplex(0.0,0.0);
				host_batchedPot[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedHff[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedHff2[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedHff3[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedS[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedS1[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedS2[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedV[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedInvV[i][j] = make_hipDoubleComplex(0.0,0.0);
				// host_batchedP[i][j] = 0;
			}
		}
		printf("Assigning device memory for Non Standard Interaction.\n");
		// double* denergy = nullptr;
		// hipMalloc(reinterpret_cast<void **>(&denergy), _batch_count * sizeof(double));

		for (int i = 0; i < _batch_count; i++) {
			hipMalloc(reinterpret_cast<void **>(&device_batchedU[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedDM[i]),		sizeof(data_type) * m*n);
			hipMalloc(reinterpret_cast<void **>(&device_batchedPot[i]),	sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedHff[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedHff2[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedHff3[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedS[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedS1[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedS2[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedV[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedInvV[i]), sizeof(data_type) * m*n);
			// hipMalloc(reinterpret_cast<void **>(&device_batchedP[i]), sizeof(double) * m*n));
		}

		hipMalloc(reinterpret_cast<void **>(&batchedU),  	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedDM),  	_batch_count*sizeof(data_type *));
		hipMalloc(reinterpret_cast<void **>(&batchedPot), 	_batch_count*sizeof(data_type *));
		// hipMalloc(reinterpret_cast<void **>(&batchedHff), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedHff2), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedHff3), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedS), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedS1), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedS2), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedV), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedInvV), 	_batch_count*sizeof(data_type *)));
		// hipMalloc(reinterpret_cast<void **>(&batchedP), 	_batch_count*sizeof(double *)));
		printf("Copying host memory to device memory.\n");
		for (int i = 0; i < _batch_count; i++) {
			// hipMemcpy(device_batchedU[i], host_batchedU[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedDM[i], host_batchedDM[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			hipMemcpy(device_batchedPot[i], host_batchedPot[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice);
			// hipMemcpy(device_batchedHff[i], host_batchedHff[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedHff2[i], host_batchedHff2[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedHff3[i], host_batchedHff3[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedS[i], host_batchedS[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedS1[i], host_batchedS1[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedS2[i], host_batchedS2[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedV[i], host_batchedV[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedInvV[i], host_batchedInvV[i], sizeof(data_type) * m*n, hipMemcpyHostToDevice));
			// hipMemcpy(device_batchedP[i], host_batchedP[i], sizeof(double) * m*n, hipMemcpyHostToDevice));
		}
		hipMemcpy(batchedU, device_batchedU, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedDM, device_batchedDM, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		hipMemcpy(batchedPot, device_batchedPot, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedHff, device_batchedHff, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedHff2, device_batchedHff2, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedHff3, device_batchedHff3, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedS, device_batchedS, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedS1, device_batchedS1, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedS2, device_batchedS2, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedV, device_batchedV, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedInvV, device_batchedInvV, sizeof(data_type*)*_batch_count, hipMemcpyHostToDevice);
		// hipMemcpy(batchedP, device_batchedP, sizeof(double*)*_batch_count, hipMemcpyHostToDevice);
		// clock_t start_time = clock();
		printf("Calling NSI Kernel with %d blocks and %.0f threads per block\n", blocks, threads);
		gpu_nsi<<<blocks, threads>>>(
			_U, _batch_count, _sigN, _L, _rho, _dm, _alpha, _events,
			batchedDM, batchedPot, batchedU);
		hipDeviceSynchronize();
		// // CuBlas Operations
		// hipblasHandle_t cublasH = NULL;
		// // hipStream_t stream = NULL;

		// const data_type alpha = {1.0, 0.0};
		// const data_type beta = {1.0, 0.0};

		// CUBLAS_CHECK(hipblasCreate(&cublasH));
		// // hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
		// // CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
		// CUBLAS_CHECK(hipblasZgemmBatched(
		// 	cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedU, lda,
		// 	(const data_type*  const*) batchedDM, ldb, &beta, batchedHff, ldc, _batch_count));
		// // hipStreamSynchronize(stream));
		// CUBLAS_CHECK(hipblasZgemmBatched(
		// 	cublasH, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, &alpha, (const data_type * const *)batchedHff, lda,
		// 	(const data_type*  const*) batchedU, ldb, &beta, batchedHff2, ldc, _batch_count));
		// // hipStreamSynchronize(stream));
		// // CUBLAS_CHECK(hipblasDestroy(cublasH));

		// // Calculating eigen value with cuSolver
		// hipsolverHandle_t cusolverH = NULL;
		// hipsolverGesvdjInfo_t gesvdj_params = NULL;
		// data_type *d_A = nullptr;    /* lda-by-m-by-batchSize */
		// data_type *h_A = (data_type*) malloc(sizeof(data_type) * _batch_count * m * n);    /* lda-by-m-by-batchSize */
		// data_type *d_U = nullptr;    /* lda-by-m-by-batchSize */
		// data_type *h_U = (data_type*) malloc( sizeof(data_type) * ldu * m * _batch_count);    /* lda-by-m-by-_batch_count */
		// data_type *d_V = nullptr;    /* lda-by-m-by-_batch_count */
		// data_type *h_V = (data_type*) malloc( sizeof(data_type) * ldv * n * _batch_count);    /* lda-by-m-by-_batch_count */
		// double* S = (double*) malloc(sizeof(hipDoubleComplex) * minmn * _batch_count);
		// double *d_S = nullptr; /* minmn-by-batchSize */
		// int* info = (int*) malloc(sizeof(int) * _batch_count);
		// int *d_info = nullptr; /* batchSize */


		// int lwork = 0;            /* size of workspace */
		// data_type *d_work = nullptr; /* device workspace for getrf */

		// // const double tol = 1.e-7;
		// // const int max_sweeps = 50;
		// const int sort_svd = 0;                                  /* don't sort singular values */
		// const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */
		// hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * _batch_count * m * n));
		// sum_batched<<<blocks, threads>>> (batchedPot, batchedHff2, batchedHff3, d_A, _batch_count);
		// hipDeviceSynchronize();
		// // Hff[0] = U[0] * DM[0] * UC[0] + Pot[0]
		// // Hff[1] = U[1] * DM[1] * UC[1] + Pot[1]
		// // Hff[N-1] = U[N-1] * DM[N-1] * UC[N-1] + Pot[N-1]
		// /* step 1: create cusolver handle, bind a stream */
		// hipsolverDnCreate(&cusolverH);
		// // hipsolverSetStream(cusolverH, stream);

		// /* step 2: configuration of syevj */
		// hipsolverDnCreateGesvdjInfo(&gesvdj_params);

		// /* default value of tolerance is machine zero */
		// // hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol);

		// /* default value of max. sweeps is 100 */
		// // hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);


		// /* disable sorting */
		// hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd);
		// hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(data_type) * ldu * m * _batch_count));
		// hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(data_type) * ldv * n * _batch_count));
		// hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * minmn * _batch_count));
		// hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * _batch_count));

		// hipsolverDnZgesvdjBatched_bufferSize(cusolverH, jobz, m, n, d_A, lda, d_S, d_U,
		// 	ldu, d_V, ldv, &lwork, gesvdj_params, _batch_count);
		// hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * lwork));

		// hipsolverDnZgesvdjBatched(cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V,
		// 	ldv, d_work, lwork, d_info, gesvdj_params, _batch_count);

		// // building S from eigenvalues and ordering V to array of vectors
		// building_SandV<<<blocks, threads>>> (
		// 	batchedS, batchedDM, d_S, d_V, batchedV, _batch_count,
		// 	make_hipDoubleComplex(ProbConst::I.real(), ProbConst::I.imag()),
		// 	ProbConst::GevkmToevsq, _L);
		// // building the inverse of V from batchedV
		// CUBLAS_CHECK(
		// 	cublasZmatinvBatched(
		// 		cublasH, n, (const hipDoubleComplex * const *) batchedV, lda, batchedInvV, lda,
		// 		d_info, _batch_count)
		// );
		// // S = S*V * Vinv
		// CUBLAS_CHECK(hipblasZgemmBatched(
		// 	cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedV, lda,
		// 	(const data_type*  const*) batchedS, ldb, &beta, batchedS1, ldc, _batch_count));
		// CUBLAS_CHECK(hipblasZgemmBatched(
		// 	cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, (const data_type * const *)batchedS1, lda,
		// 	(const data_type*  const*) batchedInvV, ldb, &beta, batchedS2, ldc, _batch_count));
		// buildP<<<blocks, threads>>>(batchedP, batchedS2, _batch_count);
		// clock_t stop_time = clock();
		// printf("Computation time: %.7fs\n", (double)(stop_time - start_time)/CLOCKS_PER_SEC);
		for (int i = 0; i < _batch_count; i++) {
			// hipMemcpy(host_batchedU[i], device_batchedU[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedDM[i], device_batchedDM[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			hipMemcpy(host_batchedPot[i], device_batchedPot[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedHff[i], device_batchedHff[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedHff2[i], device_batchedHff2[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedHff3[i], device_batchedHff3[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedS[i], device_batchedS[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedS2[i], device_batchedS2[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedV[i], device_batchedV[i], sizeof(data_type)* m*n, hipMemcpyDeviceToHost );
			// hipMemcpy(host_batchedP[i], device_batchedP[i], sizeof(double)* m*n, hipMemcpyDeviceToHost );
		}
		// // hipMemcpy(h_A, d_A, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost));
		// hipMemcpy(h_U, d_U, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost));
		// // hipMemcpy(h_V, d_V, sizeof(data_type) * _batch_count * m * n, hipMemcpyDeviceToHost));
		// hipMemcpy(S, d_S, sizeof(double) * minmn * _batch_count, hipMemcpyDeviceToHost));
		// hipMemcpy(info, d_info, sizeof(int) * _batch_count, hipMemcpyDeviceToHost));

		// // hipStreamSynchronize(stream));
		// hipDeviceSynchronize();

		for (int i=0; i < _batch_count; i++) {
			// printf("U[%d]:\n",i);
			// print_matrix(m, n, host_batchedU[i], lda);
			printf("DM[%d]:\n",i);
			print_matrix(m, n, host_batchedDM[i], lda);
			printf("Pot[%d]:\n",i);
			print_matrix(m, n, host_batchedPot[i], lda);
		// 	printf("Hff[%d]:\n",i);
		// 	print_matrix(m, n, host_batchedHff[i], lda);
		// 	printf("Hff2[%d]:\n",i);
		// 	print_matrix(m, n, host_batchedHff2[i], lda);
		// 	printf("Hff3[%d]:\n",i);
		// 	print_matrix(m, n, host_batchedHff3[i], lda);
		// 	// printf("H_A[%d]:\n",i);
		// 	// print_matrix(m, n, h_A +  m * lda * i , 3);
		// 	std::printf("Eigen Values: \n");
		// 	for (int v = 0; v < minmn; v++) {
		// 		std::printf("S0(%d) = %e\n", v + 1, S[i * m + v]);
			// }
		// 	printf("Eigen Vectors:\n");
		// 	print_matrix(m, m, h_V + i * m * lda, ldv);
		// 	printf("S[%d]:\n", i);
		// 	print_matrix(m, m, host_batchedS[i], lda);
		// 	// printf("V[%d]:\n", i);
		// 	// print_matrix(m, m, host_batchedV[i], lda);
		// 	printf("S2[%d]:\n", i);
		// 	print_matrix(m, m, host_batchedS2[i], lda);
		// 	printf("P[%d]:\n", i);
		// 	print_matrix(m, m, host_batchedP[i], lda);
		}
		for (int i = 0; i < _batch_count; i++) {
			// hipHostFree(host_batchedU[i]);
			hipHostFree(host_batchedDM[i]);
			hipHostFree(host_batchedPot[i]);
			// hipHostFree(host_batchedHff[i]);
			// hipHostFree(host_batchedHff2[i]);
			// hipHostFree(host_batchedHff3[i]);
			// hipHostFree(host_batchedS[i]);
			// hipHostFree(host_batchedS1[i]);
			// hipHostFree(host_batchedS2[i]);
			// hipHostFree(host_batchedV[i]);
			// hipHostFree(host_batchedInvV[i]);
			// hipHostFree(host_batchedP[i]);
			hipFree(device_batchedU[i]); 
			hipFree(device_batchedDM[i]);
			hipFree(device_batchedPot[i]);
			// hipFree(device_batchedHff[i]));
			// hipFree(device_batchedHff2[i]));
			// hipFree(device_batchedHff3[i]));
			// hipFree(device_batchedS[i])); 
			// hipFree(device_batchedS1[i]));
			// hipFree(device_batchedS2[i]));
			// hipFree(device_batchedV[i]));
			// hipFree(device_batchedInvV[i]));
			// hipFree(device_batchedP[i]));
		}
		hipFree(batchedU);
		hipFree(batchedDM);
		hipFree(batchedPot);
		// hipFree(batchedHff));
		// hipFree(batchedHff2));
		// hipFree(batchedHff3));
		// hipFree(batchedS));
		// hipFree(batchedS1));
		// hipFree(batchedS2));
		// hipFree(batchedV));
		// hipFree(batchedInvV));
		// hipFree(batchedP));

		
		// hipFree(d_A));
		// hipFree(d_U));
		// hipFree(d_V));
		// hipFree(d_S));
		// hipFree(d_info));
		// hipFree(d_work));

		// hipsolverDnDestroyGesvdjInfo(gesvdj_params);
		// hipsolverDnDestroy(cusolverH);
		// hipStreamDestroy(stream));
		hipDeviceReset();
}

