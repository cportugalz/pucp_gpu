#include "hip/hip_runtime.h"
#include "eigenv.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdio>


__global__ void eigenv(int _batch, hipDoubleComplex* _dinput){
    int tidx = blockDim.x*blockIdx.x + threadIdx.x;
    if (tidx < _batch){
        printf("%e ", _dinput[tidx].x);
    }

}

void eigen_system(int _batch, hipDoubleComplex* _input, double _error){
    using data_type =  hipDoubleComplex;
    data_type* dinput = nullptr;
    // data_type* dbatched[_batch];
    hipMalloc(reinterpret_cast<void **>(&dinput), 9*sizeof(data_type));
    hipMemcpy(dinput, _input, 9*sizeof(data_type), hipMemcpyHostToDevice);
    float threads = 1024;
    int blocks = ceil(_batch/threads);
    eigenv<<< blocks, threads>>>(_batch, dinput);
    hipFree(dinput);
}
